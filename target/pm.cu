// 在 GPU 上进行 Pattern Matching.
#include <algorithm>

#include "consts/project_consts.hpp"
#include "core/schedule.hpp"
#include "engine/engine.cuh"
#include "infra/graph_backend.cuh"
#include "utils/cuda_utils.cuh"
#include "utils/utils.hpp"

// generated config
#include "generated/default_config.hpp"

int main(int argc, char* argv[]) {
    if (argc < 3) {
        throw std::runtime_error(
            "Usage: ./pm <graph_name> <pattern_str> [hash_code]");
    }
    std::string graph_name{argv[1]};
    std::cout << "Graph Name: " << graph_name << std::endl;
    // 数据图
    std::ifstream graph_file{PROJECT_ROOT / graph_name};
    if (!graph_file.is_open()) {
        throw std::runtime_error("Cannot open the graph file");
    }
    // 模式图
    std::string pattern_str{argv[2]};

    // 标识配置的哈希码
    std::string hash_code = "";
    if (argc > 3) {
        hash_code = argv[3];
        std::cout << "Hash Code: " << hash_code << std::endl;
    }

    // 1. 构建 Context
    // Schedule
    Core::Schedule schedule{pattern_str};
    schedule.output();
    // 图后端
    Infra::GlobalMemoryGraph<default_config> graph{graph_file, true};
    graph.output();
    int set_size = graph.max_degree();

    // 设备上下文
    Engine::DeviceContext<default_config> context{schedule, graph};
    std::cout << "Size of Device Context: " << sizeof(context) << " Bytes"
              << std::endl;

    context.to_device();

    // 2. 构建 Engine
    Engine::Executor<default_config> engine{set_size, GPU_DEVICE};
    std::cout << "Size of Executor: " << sizeof(engine) << " Bytes"
              << std::endl;

    // 3. 进行 Match
    std::cout << "Enter the Search..." << std::endl;

    auto time_start = std::chrono::high_resolution_clock::now();

    long long ans = engine.perform_search(context);

    auto time_end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(
                        time_end - time_start)
                        .count();

    std::cout << "Time: " << duration << " us (" << double(duration) / 1000000
              << " s)" << std::endl;

    // 4. 输出结果

    long long total_count = 0;
    gpuErrchk(
        hipMemcpyFromSymbol(&total_count, HIP_SYMBOL(GPU::counter), sizeof(long long)));

    std::cout << "(Total Intersection Count: " << total_count << ")"
              << std::endl;

    std::cout << "Answer: " << ans << std::endl;

    // 5. 给 tuning 输出的

    if (hash_code != "") {
        output_result_files(hash_code, double(duration) / 1000000, ans);
    }
    return 0;
}